#include "hip/hip_runtime.h"
// Ingemars rewrite of the julia demo, integrating the OpenGL parts.
// The CUDA parts are - intentionally - NOT rewritten, and have some
// serious performance problems. Find the problems and make this a�
// decently performing CUDA program.

// Compile with
// nvcc -lglut -lGL interactiveJulia.cu -o interactiveJulia

#include <GL/glut.h>
#include <GL/gl.h>
#include <stdio.h>

// Image data
	unsigned char	*pixels;
	int	 gImageWidth, gImageHeight;

// Init image data
void initBitmap(int width, int height)
{
	pixels = (unsigned char *)malloc(width * height * 4);
	gImageWidth = width;
	gImageHeight = height;
}

#define DIM 1024

// Complex number class
struct hipComplex
{
    float   r;
    float   i;
    
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    
    __device__ float magnitude2( void )
    {
        return r * r + i * i;
    }
    
    __device__ hipComplex operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    
    __device__ hipComplex operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, float r, float im)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

//    hipComplex c(-0.8, 0.156);
    hipComplex c(r, im);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return i;
    }

    return i;
}

__global__ void kernel( unsigned char *ptr, float r, float im)
{
    // map from blockIdx to pixel position
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int offset = x + y * DIM;

    // now calculate the value at that position
    int juliaValue = julia( x, y, r, im );
    ptr[offset*4 + 0] = 255 * juliaValue/200;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

float theReal, theImag;

// Compute CUDA kernel and display image
void Draw()
{
	unsigned char *dev_bitmap;
  hipEvent_t startEvent, endEvent;
  float time;
	
	hipMalloc( &dev_bitmap, gImageWidth*gImageHeight*4 );

 // Insert event before kernel has run
  hipEventCreate(&startEvent);
  hipEventRecord(startEvent);

  dim3 block(1, 1);
	dim3	grid(DIM,DIM);
	kernel<<<grid,block>>>( dev_bitmap, theReal, theImag);

  // Insert event after kernel has run
  hipEventCreate(&endEvent);
  hipEventRecord(endEvent);

  // Wait for event to finish
  hipEventSynchronize(endEvent);

	hipDeviceSynchronize();
	hipMemcpy( pixels, dev_bitmap, gImageWidth*gImageHeight*4, hipMemcpyDeviceToHost );
	
	hipFree( dev_bitmap );
	
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glDrawPixels( gImageWidth, gImageHeight, GL_RGBA, GL_UNSIGNED_BYTE, pixels );
	glutSwapBuffers();

  // Print 
  hipEventElapsedTime(&time, startEvent, endEvent);
  printf("Kernel time: %f milliseconds\n", time);

  hipEventDestroy(startEvent);
  hipEventDestroy(endEvent);
}

void MouseMovedProc(int x, int y)
{
	theReal = -0.5 + (float)(x-400) / 500.0;
	theImag = -0.5 + (float)(y-400) / 500.0;
	  printf("real = %f, imag = %f\n", theReal, theImag);
	glutPostRedisplay ();
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
	glutInitWindowSize( DIM, DIM );
	glutCreateWindow("CUDA on live GL");
	glutDisplayFunc(Draw);
	glutPassiveMotionFunc(MouseMovedProc);
	
	initBitmap(DIM, DIM);
	
	glutMainLoop();
}
