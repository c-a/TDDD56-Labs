#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

static const int N = 2048;
static const int block_xDim = 256;
static const int block_yDim = 1;

__global__ 
void matrix_add(float* a, float* b, float *c) 
{
  uint i = (blockIdx.y * blockDim.y) + threadIdx.y;
  uint j = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (i < N && j < N) {
    uint idx = i * N + j;
  	c[idx] = a[idx] + b[idx];
  }
}

int main()
{
	const int size = N*N*sizeof(float);

  hipEvent_t startEvent, endEvent;
  float time;

	float* a = new float[N*N];
	float* b = new float[N*N];
  float* c = new float[N*N];
	float *ad, *bd, *cd;

  // Initialize a and b
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	
	hipMalloc( (void**)&ad, size );
  hipMalloc((void**)&bd, size);
  hipMalloc((void**)&cd, size);

  /* Upload a to ad and b to bd */
  hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

  // Insert event before kernel has run
  hipEventCreate(&startEvent);
  hipEventRecord(startEvent);

	dim3 dimBlock( block_xDim, block_yDim);
	dim3 dimGrid( (N + block_xDim - 1)/block_xDim, (N + block_yDim - 1)/block_yDim);
	matrix_add<<<dimGrid, dimBlock>>>(ad, bd, cd);

  // Insert event after kernel has run
  hipEventCreate(&endEvent);
  hipEventRecord(endEvent);

  // Wait for event to finish
  hipEventSynchronize(endEvent);

	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

	hipFree( ad );
	hipFree( bd );
	hipFree( cd );
#if 0
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}
#endif

  hipEventElapsedTime(&time, startEvent, endEvent);
  printf("Time: %f milliseconds\n", time);

	return EXIT_SUCCESS;
}
