#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = sqrt(c[threadIdx.x]);
}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
  int i;
	
	hipMalloc( (void**)&cd, size );

  /* Fill c with data */
  for (i = 0; i < N; i++)
    c[i] = i*i;

  /* Upload c to cd */
  hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
 
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++)
		printf("%f:%f ", c[i], sqrtf(i*i));
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
