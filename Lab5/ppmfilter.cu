#include "hip/hip_runtime.h"

#include <stdio.h>
#include "readppm.c"
#ifdef __APPLE__
	#include <GLUT/glut.h>
	#include <OpenGL/gl.h>
#else
	#include <GL/glut.h>
#endif

static const int nBlocks = 32;
static const int outputPerBlock = 512 / nBlocks;
static const int threadsPerBlock = outputPerBlock + 4;

__constant__ int kernel[5*5];

__global__ void filter(unsigned char *image, unsigned char *out, int n, int m)
{
    __shared__ unsigned char S[threadsPerBlock*threadsPerBlock*3];
	int i = blockIdx.y * outputPerBlock + (threadIdx.y - 2);
	int j = blockIdx.x * outputPerBlock + (threadIdx.x - 2);

// printf is OK under --device-emulation
//  printf("%d %d %d %d\n", i, j, n, m);

  if (i < 0 || j < 0 || i >= n || j >= m)
    return;

  //printf("%d %d\n", i, j);
  S[(threadIdx.y*threadsPerBlock + threadIdx.x)*3+0] = image[(i*n+j)*3+0];
  S[(threadIdx.y*threadsPerBlock + threadIdx.x)*3+1] = image[(i*n+j)*3+1];
  S[(threadIdx.y*threadsPerBlock + threadIdx.x)*3+2] = image[(i*n+j)*3+2];

  __syncthreads();

  if ((threadIdx.x >= 2 && threadIdx.x < (threadsPerBlock - 2)) &&
      (threadIdx.y >= 2 && threadIdx.y < (threadsPerBlock - 2)))
  {
      int y = threadIdx.y;
      int x = threadIdx.x;

      if (i <= 1 || i >= (n-2) || j <= 1 || j >= (m-2)) {
	      out[(i*n+j)*3+0] = S[(y*threadsPerBlock+x)*3+0];
	      out[(i*n+j)*3+1] = S[(y*threadsPerBlock+x)*3+1];
	      out[(i*n+j)*3+2] = S[(y*threadsPerBlock+x)*3+2];
      }
      else {
      	int sumx, sumy, sumz, k, l;
   			// Filter kernel
		    sumx=0;sumy=0;sumz=0;
		    for(k=-2;k<3;k++)
			    for(l=-2;l<3;l++)
			    {
            int kernelIndex = (k+2)*5 + (l+2);
            sumx += kernel[kernelIndex]*S[((y+k)*threadsPerBlock+(x+l))*3+0];
            sumy += kernel[kernelIndex]*S[((y+k)*threadsPerBlock+(x+l))*3+1];
            sumz += kernel[kernelIndex]*S[((y+k)*threadsPerBlock+(x+l))*3+2];
			    }

		    out[(i*n+j)*3+0] = sumx/256;
		    out[(i*n+j)*3+1] = sumy/256;
		    out[(i*n+j)*3+2] = sumz/256;
     }
  }
}


// Compute CUDA kernel and display image
void Draw()
{
	unsigned char *image, *out;
	int n, m;
	unsigned char *dev_image, *dev_out;
  hipEvent_t startEvent, endEvent;
  float time;
	
	image = readppm("maskros512.ppm", &n, &m);
	out = (unsigned char*) malloc(n*m*3);
	
	hipMalloc( (void**)&dev_image, n*m*3);
	hipMalloc( (void**)&dev_out, n*m*3);
	hipMemcpy( dev_image, image, n*m*3, hipMemcpyHostToDevice);
	
  int kernel_[] = {
    1,  4,  6,  4, 1,
    4, 16, 24, 16, 4,
    6, 24, 36, 24, 6,
    4, 16, 24, 16, 4,
    1,  4,  6,  4, 1
  };

  hipMemcpyToSymbol(HIP_SYMBOL(kernel), kernel_, 5*5*sizeof(int)); 

	dim3 dimBlock( threadsPerBlock, threadsPerBlock );
	dim3 dimGrid( nBlocks, nBlocks );
	
  // Insert event before kernel has run
  hipEventCreate(&startEvent);
  hipEventRecord(startEvent);

	filter<<<dimGrid, dimBlock>>>(dev_image, dev_out, n, m);

  // Insert event after kernel has run
  hipEventCreate(&endEvent);
  hipEventRecord(endEvent);

  // Wait for event to finish
  hipEventSynchronize(endEvent);

	hipDeviceSynchronize();
	
	hipMemcpy( out, dev_out, n*m*3, hipMemcpyDeviceToHost );
	hipFree(dev_image);
	hipFree(dev_out);
	
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glRasterPos2f(-1, -1);
	glDrawPixels( n, m, GL_RGB, GL_UNSIGNED_BYTE, image );
	glRasterPos2i(0, -1);
	glDrawPixels( n, m, GL_RGB, GL_UNSIGNED_BYTE, out );
	glFlush();

  hipEventElapsedTime(&time, startEvent, endEvent);
  printf("Time: %f milliseconds\n", time);
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );
	glutInitWindowSize( 1024, 512 );
	glutCreateWindow("CUDA on live GL");
	glutDisplayFunc(Draw);
	
	glutMainLoop();
}
